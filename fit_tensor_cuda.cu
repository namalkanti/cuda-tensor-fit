#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "fit_tensor.h"

//clones double array and copies to gpu
double* cuda_double_copy(double* arr, size_t len){
    double* carr;
    hipMalloc(&carr, sizeof(double) * len);
    hipMemcpy(carr, arr, sizeof(double) * len, hipMemcpyHostToDevice);
    return carr;
}

//clones double array and copies to host 
double* cuda_double_return(double* carr, size_t len){
    double* arr = (double*) malloc(sizeof(double) * len);
    hipMemcpy(arr, carr, sizeof(double) * len, hipMemcpyDeviceToHost);
    return arr;
}

//allocates space for a double array on the device
void cudal_double_alloc(double* ptr, int len){
    hipMalloc(&ptr, len);
}

//frees double device memory
void free_cuda(double* ptr){
    hipFree(ptr);
}

//kernel to take entire array and run cutoff log function
__global__ void cutoff_log_kernel(double* input, double* output, double min_signal, size_t len){
    int tid = blockIdx.x;
    if (input[tid] < min_signal){
        output[tid] = log(min_signal);
    }
    else{
        output[tid] = log(input[tid]);
    }
}

//kernel to take entire array and exp it
__global__ void exp_kernel(double* input, double* output, size_t len){
    int tid = blockIdx.x;
    output[tid] = pow(M_E, input[tid]);
}

//function that take in a complete signal matrix and fits it, cuda version
void fit_complete_signal(matrix* ols_fit, matrix* design_matrix, matrix* signal, double min_signal, double min_diffusivity, tensor** tensor_output){
    return;
}
