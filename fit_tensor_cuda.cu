#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "fit_tensor.h"

//clones double array and copies to gpu
double* cuda_double_copy(double* arr, size_t len){
    double* carr;
    hipMalloc(&carr, sizeof(double) * len);
    hipMemcpy(carr, arr, sizeof(double) * len, hipMemcpyHostToDevice);
    return carr;
}

//clones double array and copies to host 
double* cuda_double_return(double* carr, size_t len){
    double* arr = (double *) malloc(sizeof(double) * len);
    hipMemcpy(arr, carr, sizeof(double) * len, hipMemcpyDeviceToHost);
    return arr;
}

//allocates space for a double array on the device
void cuda_double_alloc(double* ptr, int len){
    hipMalloc(&ptr, len);
}

//frees double device memory
void free_cuda(double* ptr){
    hipFree(ptr);
}

//kernel to take entire array and run cutoff log function
__global__ void cutoff_log_kernel(double* input, double* output, double min_signal){
    int tid = blockIdx.x;
    if (input[tid] < min_signal){
        output[tid] = log(min_signal);
    }
    else{
        output[tid] = log(input[tid]);
    }
}

//Function to launch cutoff log kernel
void cutoff_log_cuda(double* input, double* output, double min_signal, int block_grid_rows){
  cutoff_log_kernel<<<block_grid_rows, 1>>>(input, output, min_signal);
}

//kernel to take entire array and exp it
__global__ void exp_kernel(double* input, double* output){
    int tid = blockIdx.x;
    output[tid] = pow(M_E, input[tid]);
}


//Kernel catapult
void exp_cuda(double* input, double* output, int block_grid_rows){
  exp_kernel<<<block_grid_rows, 1>>>(input, output);
}

//function that take in a complete signal matrix and fits it, cuda version
void fit_complete_signal(matrix* ols_fit, matrix* design_matrix, matrix* signal, double min_signal, double min_diffusivity, tensor** tensor_output){
    return;
}
