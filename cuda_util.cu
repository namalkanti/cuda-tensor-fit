#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <hipblas.h>
extern "C" {
#include "cuda_util.h"
}
#include "BatchedSolver/solve.h"

#define IDX2C(i, j, ld) ((j)*(ld)+(i))

#define TENSOR_DIMENSIONS 3
#define TENSOR_INPUT_ELEMENTS 6
#define TENSOR_ELEMENTS 9
#define EIGENDECOMPOSITION_ELEMENTS 12


//Helper function declarations
double* convert_matrix_to_fortran_and_load_to_gpu(matrix const* mat);
void get_matrix_from_gpu_and_convert_from_fortran(double const* gpu_pointer, matrix* mat);

//Kernel declarations
__global__ void cutoff_log_kernel(double* device_array, double min_signal);
__global__ void exp_kernel(double* cuda_array);
__global__ void weighting_kernel (double* matrices, double* weights, double* results); 
__global__ void weighting_kernel_transposed(double* matrices, double* weights, double* results); 
__global__ void transpose_kernel(double const* matrices, double* transposed);
__global__ void assemble_tensors(double const* tensor_input, double* tensors);
__global__ void eigenvalue_kernel(double* data, double* eigenvalues);
__global__ void eigenvector_kernel(double const* data, double* eigenvectors);

//Device function declarations
__device__ double trace(double const* data, double* eigenvectors);
__device__ double determinant(double const* data, int offset);
__device__ double diagonal_q_difference(double const* data, int offset, double q);
__device__ void subtract_from_diagonal(double* data, int offset, double value);
__device__ void scale_gpu_data_segment(double* data, int offset, double scalar);


extern "C"
matrix* process_signal(matrix const* signal, double min_signal){
    double* signal_data = array_clone(signal->data);
    int signal_length = signal->rows * signal->columns;
    double* kernel_results = cutoff_log_cuda(signal_data, min_signal, signal_length);
    double* processed_signal_data = cuda_double_copy_to_gpu(kernel_results);
    matrix* processed_signal = {processed_signal_data, signal->rows, signal->columns};
    free(signal_data);
    free_cuda_memory(kernel_results);
    return processed_signal;
}

extern "C"
matrix* generate_weights(matrix const* ols_fit_matrix, matrix const* signal){
    matrx* weights = cuda_matrix_dot(ols_fit_matrix, signal);
    double* gpu_weights_data = cuda_double_copy_to_gpu(weights->data);
    matrix gpu_weights= {.data = gpu_weights_data, .rows = weights->rows, .columns = weights->columns};
    free_matrix(weights);
    return gpu_weights;
}

extern "C"
double* cuda_fitter(matrix const* design_matrix, matrix const* weights, matrix const* signal){
    double* weighted_design_data = matrix_weigher(design_matrix->data, weights->data, data->rows, 
            data->columns, weights->rows);
    double* solution_vectors;
    int signal_elements = signal->rows * signal->columns;
    cuda_double_allocate(solution_vectors, signal_elements);
    int solver_status = dsolve_batch(weighted_design_data, signal->data, solution_vectors, 
            signal->columns, signal->rows);
    if ( 0 > solver_status) {
        fputs("Batched solver failed to run correctly, program will fail", stderr);
    }
    return solution_vectors
}

extern "C"
double* cuda_decompose_tensors(double const* tensors_input, int number_of_tensors){
    double* tensors, tensors_copy;
    cuda_double_allocate(tensors_input, TENSOR_ELEMENTS * number_of_tensors);
    dim3 grid, block;
    grid.x = number_of_tensors;
    block.x = 1;
    block.y = 1;
    assemble_tensors<<<grid, block>>>(tensors_input, tensors);
    assemble_tensors<<<grid, block>>>(tensors_input, tensors_copy
    double* eigenvalues, eigenvectors, eigendecomposition;
    eigendecomposition = malloc(sizeof(double) * EIGENDECOMPOSITION_ELEMENTS * number_of_tensors);  
    cuda_double_allocate(eigenvalues, TENSOR_DIMENSIONS * number_of_tensors);
    cuda_double_allocate(eigenvectors, TENSOR_ELEMENTS * number_of_tensors);
    eigenvalue_kernel<<<grid, block>>>(tensors_copy, eigenvalues);
    eigenvector_kernel<<<grid, block>>>(tensors, eigenvectors)
    assemble_eigendecomposition(eigenvalues, eigenvectors, eigendecomposition);
    return eigendecomposition;
}

extern "C"
matrix* process_matrix(matrix const* design_matrix){
    double* gpu_matrix_data = convert_matrix_to_fortran_and_load_to_gpu(design_matrix);
    matrix* processed_matrix = {gpu_matrix_data, design_matrix->rows, design_matrix->columns};
    return processed_matrix;
}

extern "C"
void assemble_eigendecomposition(double const* eigenvalues, double const* eigenvectors, double* eigendecomposition){
}

extern "C"
double* cuda_double_copy_to_gpu(double const* local_array, int array_length){
    double* cuda_array;
    hipMalloc(&cuda_array, sizeof(double) * array_length);
    hipMemcpy(cuda_array, local_array, sizeof(double) * array_length, hipMemcpyHostToDevice);
    return cuda_array;
}

extern "C"
double* cuda_double_return_from_gpu(double const* cuda_array, int array_length){
    double* result_array = (double *) malloc(sizeof(double) * array_length);
    hipMemcpy(result_array, cuda_array, sizeof(double) * array_length, hipMemcpyDeviceToHost);
    return result_array;
}

extern "C"
void cuda_double_allocate(double* pointer, int pointer_length){
    hipMalloc(&pointer, pointer_length);
    hipMemset(&pointer, 0, pointer_length);
}

extern "C"
void free_cuda_memory(double* pointer){
    hipFree(pointer);
}

extern "C"
void free_matrix_with_cuda_pointer(matrix* gpu_matrix){
    free_cuda_memory(gpu_matrix->data);
    free(gpu_matrix);
}

extern "C"
double* cutoff_log_cuda(double const* input, double min_signal, int array_length){
    padded_array* padded_arr = pad_array(input, array_length, WARP_SIZE);
    double* device_array = cuda_double_copy_to_gpu(padded_arr->values, padded_arr->current_length);
    int blocks_in_grid = padded_arr->current_length / WARP_SIZE;
    cutoff_log_kernel<<<blocks_in_grid, WARP_SIZE>>>(device_array, min_signal);
    padded_arr->values = cuda_double_return_from_gpu(device_array, padded_arr->current_length);
    double* result_array = get_array_from_padded_array(padded_arr);
    free_cuda_memory(device_array);
    free_padded_array(padded_arr);
    return result_array;
}

extern "C"
double* exp_cuda(double const* input, int array_length){
    padded_array* padded_arr = pad_array(input, array_length, WARP_SIZE);
    double* device_array = cuda_double_copy_to_gpu(padded_arr->values, padded_arr->current_length);
    int blocks_in_grid = padded_arr->current_length/ WARP_SIZE;
    exp_kernel<<<blocks_in_grid, WARP_SIZE>>>(device_array);
    padded_arr->values = cuda_double_return_from_gpu(device_array, padded_arr->current_length);
    double* output_array = get_array_from_padded_array(padded_arr);
    free_cuda_memory(device_array);
    free_padded_array(padded_arr);
    return output_array;
}

extern "C"
matrix* cuda_matrix_dot(matrix const* matrix1, matrix const* matrix2){
    hipblasStatus_t status;
    hipblasHandle_t handle;
    status = hipblasCreate(&handle);
    if ( status != HIPBLAS_STATUS_SUCCESS ) {
        puts("Failed to retrieve cublas handle.");
    }
    double* gpu_array1 = convert_matrix_to_fortran_and_load_to_gpu(matrix1);
    double* gpu_array2 = convert_matrix_to_fortran_and_load_to_gpu(matrix2);
    double* gpu_output;
    hipMalloc(&gpu_output, sizeof(double)* matrix1->rows * matrix2->columns);
    const double alpha = 1;
    const double beta = 0;
    status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix1->rows, matrix2->columns, matrix1->columns, 
            &alpha, gpu_array1, matrix1->rows, gpu_array2, matrix2->rows, &beta, gpu_output, matrix1->rows);
    if ( status != HIPBLAS_STATUS_SUCCESS ) {
        puts("Call to cublas function failed.");
    }
    matrix* result_matrix = (matrix*) malloc(sizeof(matrix));
    double* result_matrix_data = (double*) malloc(sizeof(double) * matrix1->rows * matrix2->columns);
    result_matrix->rows = matrix1->rows;
    result_matrix->columns = matrix2->columns;
    result_matrix->data = result_matrix_data;
    get_matrix_from_gpu_and_convert_from_fortran(gpu_output, result_matrix);
    hipFree(gpu_array1);
    hipFree(gpu_array2);
    return result_matrix;
}
    
extern "C"
double* matrix_weighter (double const* matrix, double const* weights, int rows, int columns, int length, bool trans) {
    dim3 grid, block;
    int weight_length;
    grid.x = length;
    block.x = columns;
    block.y = rows;
    if ( false == trans ) {
        weight_length = columns;
    }
    else {
        weight_length = rows;
    }
    double* gpu_matrix = cuda_double_copy_to_gpu(matrix, rows * columns);
    double* gpu_weights = cuda_double_copy_to_gpu(weights, weight_length * length);
    double* gpu_results;
    hipMalloc(&gpu_results, sizeof(double) * rows * columns * length);
    if (false == trans){
        weighting_kernel<<<grid, block>>>(gpu_matrix, gpu_weights, gpu_results);
    }
    else {
        weighting_kernel_transposed<<<grid, block>>>(gpu_matrix, gpu_weights, gpu_results);
    }
    double* weighted_matrices = malloc(sizeof(double) * rows * columns * length);
    hipMemcpy(weighted_matrices, gpu_results, sizeof(double) * rows * columns * length, hipMemcpyDeviceToHost);
    hipFree(gpu_matrix);
    hipFree(gpu_weights);
    hipFree(gpu_results);
    return weighted_matrices;
}

extern "C"
double* transpose_matrices(double* matrices, int rows, int columns, int length){
    double* transposed = malloc(sizeof(double) * rows * columns * length);
    double* gpu_matrices = cuda_double_copy_to_gpu(matrices, rows * columns * length);
    double* gpu_tranposed = cuda_double_copy_to_gpu(transposed, rows * columns * length);
    dim3 grid, block;
    grid.x = length;
    block.x = columns;
    block.y = rows;
    transpose_kernel<<<grid, block>>>(double const* gpu_matrices, double* gpu_transposed);
    transposed = cuda_double_return_from_gpu(gpu_transposed, rows * columns * length);
    free_cuda_memory(gpu_matrices);
    free_cuda_memory(gpu_transposed);
    return transposed;
}

extern "C"
double* dot_matrices(double const* matrix_batch_one, int rows, double const* matrix_batch_two, int columns,
        int k, int length){
    hipblasStatus_t status;
    hipblasHandle_t handle;
    status = hipblasCreate(&handle);
    if ( status != HIPBLAS_STATUS_SUCCESS ) {
        puts("Failed to retrieve cublas handle.");
    }
    double* transposed_batch1 = transpose_matrices(matrix_batch_one, rows, k, length);
    double* transposed_batch2 = transpose_matrices(matrix_batch_two, k, columns, length);
    double* gpu_array1 = cuda_double_copy_to_gpu(transposed_batch1, rows * k * length);
    double* gpu_array2 = cuda_double_copy_to_gpu(transposed_batch2, k *  columns * length);
    double* gpu_output;
    hipMalloc(&gpu_output, sizeof(double)* transposed_batch1->rows 
            * transposed_batch2->columns * length);
    const double alpha = 1;
    const double beta = 0;
    status = hipblasDgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rows, columns, 
            k, &alpha, gpu_array1, rows, gpu_array2, k, &beta, 
            gpu_output, rows, length);
    if ( status != HIPBLAS_STATUS_SUCCESS ) {
        puts("Call to cublas function failed.");
    }
    double* results;
    hipMalloc(&gpu_output, sizeof(double) * rows * columns * length);
    results = cuda_double_return_from_gpu(gpu_output, rows * columns * length);
    results = transpose_matrices(results, rows, columns, length);
    free_cuda_memory(gpu_array1);
    free_cuda_memory(gpu_array2);
    free_cuda_memory(gpu_output);
    free(transposed_batch1);
    free(transposed_batch2);
    return results;

}

extern "C"
double* solve_matrices(){
}

extern "C"
double* cuda_fitter(matrix const* design_matrix, matrix const* column_major_weights, 
        double const* signal, int signal_length, int number_of_signals){
    int signal_elements = signal_length;
    int total_elements = signal_elements * number_of_signals;
    double* cutoff_and_logged_signal = cutoff_log_cuda(signal, min_signal, total_elements);
    matrix signal_matrix = {cutoff_and_logged_signal, signal_elements, signal_length};
    matrix* ols_signal_dot_matrix = cuda_matrix_dot(ols_matrix, &signal_matrix);
    matrix* weights = exp_cuda(ols_signal_dot_matrix, total_elements);
    matrix* weighted_matrices = matrix_weighter(signal, weights, signal_elements, signal_length, total_elements, false);
    matrix* transposed_weighted_matrices = transpose_matrices();
    matrix* column_major_data = cuda_matrix_dot(transposed_weighted_matrices, signal);
    matrix* data = transpose_matrices(column_major_data);
    matrix* weighted_fitting_matrix = dot_matrices;
    matrix* solutions = solve_matrices();
}

extern "C"
void decompose_tensors(double const* tensors, tensor** tensor_output){
}

//Helper functions

/*Converts matrix to the data format fortran uses for CUBLAS and loads to GPU
  Returns pointer to array on GPU.*/
double* convert_matrix_to_fortran_and_load_to_gpu(matrix const* mat){
    hipblasStatus_t status;
    int length = mat->rows * mat->columns;
    double* gpu_pointer; 
    double* intermediate_matrix = (double*) malloc(sizeof(double) * length);
    hipMalloc(&gpu_pointer, sizeof(double) * length);
    int i, j;
    for (i = 0; i < mat->rows; i++ ) {
        for (j = 0; j < mat->columns; j++) {
            intermediate_matrix[IDX2C(i, j, mat->rows)] = mat->data[i * mat->rows + j];
        }
    }
    status = hipblasSetMatrix(mat->rows, mat->columns, sizeof(double), intermediate_matrix, 
            mat->rows, gpu_pointer, mat->rows);
    if ( status != HIPBLAS_STATUS_SUCCESS ) {
        puts("Failed to copy matrix to memory.");
    }
    free(intermediate_matrix);
    return gpu_pointer;
}

/*Converts matrix from the format fortran uses for CUBLAS after retrieving from GPU
  Will free gpu_pointer.
  Populates a matrix object passed in.*/
void get_matrix_from_gpu_and_convert_from_fortran(double const* gpu_pointer, matrix* mat){
    hipblasStatus_t status;
    int length = mat->rows * mat->columns;
    double* intermediate_matrix = (double*) malloc(sizeof(double) * length);
    status = hipblasGetMatrix(mat->rows, mat->columns, sizeof(double), gpu_pointer, mat->rows,
            intermediate_matrix, mat->rows);
    if ( status != HIPBLAS_STATUS_SUCCESS ) {
        puts("Failed to retrieve matrix from memory.");
    }
    int i, j;
    for (i = 0; i < mat->rows; i++ ) {
        for (j = 0; j < mat->columns; j++) {
            mat->data[i * mat->rows + j] = intermediate_matrix[IDX2C(i, j, mat->rows)];
        }
    }
    free(intermediate_matrix);
}

//Kernels

//kernel to take entire array and run cutoff log function
__global__ void cutoff_log_kernel(double* device_array, double min_signal){
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (device_array[thread_id] < min_signal){
        device_array[thread_id] = logf(min_signal);
    }
    else{
        device_array[thread_id] = logf(device_array[thread_id]);
    }
}

//kernel to take entire array and exp it
__global__ void exp_kernel(double* cuda_array){
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    cuda_array[thread_id] = expf(cuda_array[thread_id]);
}

//kernel for weighting the matrix.
__global__ void weighting_kernel (double* matrix, double* weights, double* results) {
    int matrix_grid_index = blockIdx.x * blockDim.x * blockDim.y;
    int block_index = blockDim.y * threadIdx.y + threadIdx.x;
    int matrix_index = grid_index + block_index;
    int weight_index = blockIdx.x * blockDim.x + threadIdx.x; 
    results[matrix_index] = matrices[block_index] * weights[weight_index];
}

//kernel for weighting a transposed matrix.
__global__ void weighting_kernel_transposed(double* matrix, double* weights, double* results) {
    int grid_index = blockIdx.x * blockDim.x * blockDim.y;
    int block_index = blockDim.y * threadIdx.y + threadIdx.x;
    int matrix_index = grid_index + block_index;
    int weighting_index = blockIdx.x * blockDim.y + threadIdx.y; 
    results[matrix_index] = matrices[block_index] * weights[weighting_index];
}

//kernel for transposing multiple matrices.
__global__ void transpose_kernel(double const* matrices, double* transposed) {
    int matrix_offset = blockIdx.x * blockDim.x * blockDim.y;
    int matrix_index = matrix_offset + blockDim.x * threadIdx.y + threadIdx.x;
    int transpose_index = matrix_offset + IDX2C(threadIdx.x, threadIdx.y, blockDim.y);
    transposed[transpose_index] = matrices[matrix_index];
}

//kernel for arranging tensors into symmetric matrix
__global__ void assemble_tensors(double const* tensor_input, double* tensors){
    int tensor_matrix_offset = blockIdx.x * TENSOR_DIMENSIONS * TENSOR_DIMENSIONS;
    int input_matrix_offset = blockIdx.x * TENSOR_INPUT_ELEMENTS;
    tensors[tensor_matrix_offset + 0] = tensor_input[input_matrix_offset + 0];
    tensors[tensor_matrix_offset + 1] = tensor_input[input_matrix_offset + 1];
    tensors[tensor_matrix_offset + 2] = tensor_input[input_matrix_offset + 3];
    tensors[tensor_matrix_offset + 3] = tensor_input[input_matrix_offset + 1];
    tensors[tensor_matrix_offset + 4] = tensor_input[input_matrix_offset + 2];
    tensors[tensor_matrix_offset + 5] = tensor_input[input_matrix_offset + 4];
    tensors[tensor_matrix_offset + 6] = tensor_input[input_matrix_offset + 3];
    tensors[tensor_matrix_offset + 7] = tensor_input[input_matrix_offset + 4];
    tensors[tensor_matrix_offset + 8] = tensor_input[input_matrix_offset + 5];
}

//kernel for calculating eigenvalues.
__global__ void eigenvalue_kernel(double* data, double* eigenvalues){
    int data_offset = blockIdx.x * TENSOR_DIMENSIONS * TENSOR_DIMENSIONS;
    int eigen_offset = blockIdx.x * TENSOR_DIMENSIONS;

    double diagonal_detection = diagonal_q_difference(data, data_offset + (0*TENSOR_DIMENSIONS+1), 0) + 
        diagonal_q_difference(data, data_offset + (0*TENSOR_DIMENSIONS+2), 0) + 
        diagonal_q_difference(data, data_offset(1*TENSOR_DIMENSIONS+2), 0);

    int diagonal_one_offset = data_offset + 0;
    int diagonal_two_offset = data_offset + (1*TENSOR_DIMENSIONS+1);
    int diagonal_three_offset = data_offset + (2*TENSOR_DIMENSIONS+2);

    double diagonal_one = data[diagonal_one_offset];
    double diagonal_two = data[diagonal_two_offset];
    double diagonal_three = data[diagonal_three_offset];

    if (0 == diagonal_detection) {
        eigenvalues[eigen_offset + 0] = diagonal_one;
        eigenvalues[eigen_offset + 1] = diagonal_two;
        eigenvalues[eigen_offset + 2] = diagonal_three;
    }
    else {
        q = trace(data)/3;
        p2 = diagonal_q_difference(data, diagonal_one_offset, q) +
            diagonal_q_difference(data, diagonal_two_offset, q) +
            diagonal_q_difference(data, diagonal_three_offset, q) +
            2 * pi;
        p = sqrt(p2 / 6);
        subtract_for_diagonal(data, data_offset, q);
        scale_gpu_data_segment(data, data_offset, 1/p);
        r = determinant(data, data_offset) / 2
    }

    if ( r <= -1){
        phi = pi / 3;
    }
    else if (r >= 1){
        phi = 0;
    }
    else {
        phi = acos(r) / 3;
    }

    eig1 = q + 2 * p * cos(phi);
    eig3 = q + 2 * p * cos(phi + (2 * pi / 3));
    eigenvalues[eigen_offset + 1] = 3 * q - eig1 - eig3;
    eigenvalues[eigen_offset + 0] = eig1;
    eigenvalues[eigen_offset + 2] = eig3;
}

//kernel for calculating eigenvectors
__global__ void eigenvector_kernel(double const* data, double* eigenvectors){
}

//device function to calculate trace of a 3x3 matrix
__device__ double trace(double const* data, int offset){
    return data[offset + 0] + 
        data[offset + (1 * TENSOR_DIMENSIONS + 1)] + 
        data[offset + (2 * TENSOR_DIMENSIONS + 2)]
}

//device function to calculate determinant of a 3x3 matrix
__device__ double determinant(double const* data, int offset){
}

//device function to subtract q from diagonal and square
__device__ double diagonal_q_difference(double const* data, int offset, double q){
    double element = data[offset];
    return pow(element - q, 2)
}

//device function to subtract value from diagonal
__device__ void subtract_from_diagonal(double* data, int offset, double value){
    data[offset + 0 * TENSOR_DIMENSIONS + 0] = data[offset + 0 * TENSOR_DIMENSIONS * 0] - value;
    data[offset + 1 * TENSOR_DIMENSIONS + 1] = data[offset + 1 * TENSOR_DIMENSIONS + 1] - value;
    data[offset + 2 * TENSOR_DIMENSIONS + 2] = data[offset + 2 * TENSOR_DIMENSIONS + 2] - value;
}

//device function to scale a matrix
__device__ void scale_gpu_data_segment(double* data, int offset, double scalar){
    data[offset + 0] = data[offset + 0] * scalar;  
    data[offset + 1] = data[offset + 1] * scalar;  
    data[offset + 2] = data[offset + 2] * scalar;  
    data[offset + 3] = data[offset + 3] * scalar;  
    data[offset + 4] = data[offset + 4] * scalar;  
    data[offset + 5] = data[offset + 5] * scalar;  
    data[offset + 6] = data[offset + 6] * scalar;  
    data[offset + 7] = data[offset + 7] * scalar;  
    data[offset + 8] = data[offset + 8] * scalar;  
}


