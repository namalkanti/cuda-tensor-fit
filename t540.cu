
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>

#define cudacall(call) \
do \
{ \
cudaError_t err = (call); \
if(hipSuccess != err) \
{ \
fprintf(stderr,"CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
cudaDeviceReset(); \
exit(EXIT_FAILURE); \
} \
} \
while (0)

#define cublascall(call) \
do \
{ \
cublasStatus_t status = (call); \
if(HIPBLAS_STATUS_SUCCESS != status) \
{ \
fprintf(stderr,"CUBLAS Error:\nFile = %s\nLine = %d\nCode = %d\n", __FILE__, __LINE__, status); \
cudaDeviceReset(); \
exit(EXIT_FAILURE); \
} \
\
} \
while(0)


void invert(float** src, float** dst, int n, int batchSize)
{
hipblasHandle_t handle;
cublascall(hipblasCreate(&handle));

int *P, *INFO;

cudacall(hipMalloc(&P, n * batchSize * sizeof(int)));
cudacall(hipMalloc(&INFO, batchSize * sizeof(int)));

int lda = n;

float **A = (float **)malloc(batchSize*sizeof(float *));
float **A_d, *A_dflat;
cudacall(hipMalloc(&A_d,batchSize*sizeof(float *)));
cudacall(hipMalloc(&A_dflat, n*n*batchSize*sizeof(float)));
A[0] = A_dflat;
for (int i = 1; i < batchSize; i++)
A[i] = A[i-1]+(n*n);
cudacall(hipMemcpy(A_d,A,batchSize*sizeof(float *),hipMemcpyHostToDevice));
for (int i = 0; i < batchSize; i++)
cudacall(hipMemcpy(A_dflat+(i*n*n), src[i], n*n*sizeof(float), hipMemcpyHostToDevice));

cublascall(hipblasSgetrfBatched(handle,n,A_d,lda,P,INFO,batchSize));

int INFOh[batchSize];
cudacall(hipMemcpy(INFOh,INFO,batchSize*sizeof(int),hipMemcpyDeviceToHost));

for (int i = 0; i < batchSize; i++)
if(INFOh[i] != 0)
{
fprintf(stderr, "Factorization of matrix %d Failed: Matrix may be singular\n", i);
hipDeviceReset();
exit(EXIT_FAILURE);
}

float **C = (float **)malloc(batchSize*sizeof(float *));
float **C_d, *C_dflat;
cudacall(hipMalloc(&C_d,batchSize*sizeof(float *)));
cudacall(hipMalloc(&C_dflat, n*n*batchSize*sizeof(float)));
C[0] = C_dflat;
for (int i = 1; i < batchSize; i++)
C[i] = C[i-1] + (n*n);
cudacall(hipMemcpy(C_d,C,batchSize*sizeof(float *),hipMemcpyHostToDevice));
cublascall(hipblasSgetriBatched(handle,n,(const float **)A_d,lda,P,C_d,lda,INFO,batchSize));

cudacall(hipMemcpy(INFOh,INFO,batchSize*sizeof(int),hipMemcpyDeviceToHost));

for (int i = 0; i < batchSize; i++)
if(INFOh[i] != 0)
{
fprintf(stderr, "Inversion of matrix %d Failed: Matrix may be singular\n", i);
hipDeviceReset();
exit(EXIT_FAILURE);
}
for (int i = 0; i < batchSize; i++)
cudacall(hipMemcpy(dst[i], C_dflat + (i*n*n), n*n*sizeof(float), hipMemcpyDeviceToHost));
hipFree(A_d); hipFree(A_dflat); free(A);
hipFree(C_d); hipFree(C_dflat); free(C);
hipFree(P); hipFree(INFO); hipblasDestroy(handle);
}


void test_invert()
{
const int n = 3;
const int mybatch = 4;

//Random matrix with full pivots
float full_pivot[n*n] = { 0.5, 3, 4,
1, 3, 10,
4 , 9, 16 };

//Almost same as above matrix with first pivot zero
float zero_pivot[n*n] = { 0, 3, 4,
1, 3, 10,
4 , 9, 16 };

float another_zero_pivot[n*n] = { 0, 3, 4,
1, 5, 6,
9, 8, 2 };

float another_full_pivot[n * n] = { 22, 3, 4,
1, 5, 6,
9, 8, 2 };

float *result_flat = (float *)malloc(mybatch*n*n*sizeof(float));
float **results = (float **)malloc(mybatch*sizeof(float *));
for (int i = 0; i < mybatch; i++)
results[i] = result_flat + (i*n*n);
float **inputs = (float **)malloc(mybatch*sizeof(float *));
inputs[0] = zero_pivot;
inputs[1] = full_pivot;
inputs[2] = another_zero_pivot;
inputs[3] = another_full_pivot;

for (int qq = 0; qq < mybatch; qq++){
fprintf(stdout, "Input %d:\n\n", qq);
for(int i=0; i<n; i++)
{
for(int j=0; j<n; j++)
fprintf(stdout,"%f\t",inputs[qq][i*n+j]);
fprintf(stdout,"\n");
}
}
fprintf(stdout,"\n\n");

invert(inputs, results, n, mybatch);

for (int qq = 0; qq < mybatch; qq++){
fprintf(stdout, "Inverse %d:\n\n", qq);
for(int i=0; i<n; i++)
{
for(int j=0; j<n; j++)
fprintf(stdout,"%f\t",results[qq][i*n+j]);
fprintf(stdout,"\n");
}
}
}

int main()
{
test_invert();

return 0;
}
